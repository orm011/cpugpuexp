#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <chrono>
#include <functional>
#include <hip/hip_runtime_api.h>
#include <cassert>

inline int xorshift_hash(int x) {
    x ^= x >> 12; // a
    x ^= x << 25; // b
    x ^= x >> 27; // c
    return ((unsigned int)x) * 213338717U;
}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// __global__ void
// vectorAdd(const float *A, const float *B, float *C, int numElements)
// {
//     int i = blockDim.x * blockIdx.x + threadIdx.x;

//     if (i < numElements)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

__global__ void
vectorGather(const int *index_col, const int *dimension_col, int *output, int idx_len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      output[i] = dimension_col[index_col[i]];
    }
}


__global__ void
vectorGatherNoScan(const int *dimension_col, int *output, int idx_len, int mask)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      auto x = i;
      x ^= x >> 12; // a
      x ^= x << 25; // b
      x ^= x >> 27; // c
      x = ((unsigned int)x) * 213338717U;
      auto idx = x & mask;
      output[i] = dimension_col[idx];
    }
}

#define cudaCheckErrors($call)                     \
    do { \
      hipError_t err = hipGetLastError(); \
      if (err != hipSuccess){\
        fprintf(stderr, "Error already set before call: (%s at %s:%d)\n", \
                hipGetErrorString(err),                       \
                __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
      }\
      $call;                                  \
      err = hipGetLastError(); \
      if (err != hipSuccess) { \
        fprintf(stderr, "Fatal error: (%s at %s:%d)\n", \
                hipGetErrorString(err),                \
                __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
      } \
    } while (0)


/**
 * Host main routine
 */
int
main(void)
{
  printf("FYI:\ncuda cpuDeviceId: %d\n", hipCpuDeviceId);
  auto dev = 0;
  cudaCheckErrors(hipSetDevice(dev));
  hipDeviceProp_t deviceProp;
  cudaCheckErrors(hipGetDeviceProperties(&deviceProp, dev));

  printf("some device %d properties:\n",dev);
  printf("concurrent kernels %d\n",deviceProp.concurrentKernels);
  printf("device overlap %d\n",deviceProp.deviceOverlap);
  printf("max threads per block %d\n",deviceProp.maxThreadsPerBlock);
  printf("warp size %d\n",deviceProp.warpSize);
  printf("regs per block %d\n",deviceProp.regsPerBlock);
  
  // TODO: enable this in order to try mapped memory (vs streaming)
  // hipSetDeviceFlags(hipDeviceMapHost);
  
    // Print the vector length to be used, and compute its size
    const int G = 30;
    const int M = 20;
    //const int K = 10;
    
    size_t idx_size = 1U << G;
    size_t dim_size = 256U << M;
    
    size_t idx_num = idx_size / sizeof(int);
    size_t dim_num = dim_size / sizeof(int);
    
    printf("[Gather of %lu indices into a table of %lu locations]\n", idx_num, dim_num);

    // Allocate the host input vector A
    int *h_A = nullptr;
    cudaCheckErrors(hipMallocManaged(&h_A, idx_size));

    // Allocate the host input vector B
    int *h_B = nullptr;
    hipMallocManaged(&h_B, dim_size);
    cudaCheckErrors();
    
    // Allocate the host output vector C
    int *h_C = nullptr;
    cudaCheckErrors(hipMallocManaged(&h_C, idx_size));

    int sm = __builtin_popcountl (dim_num);
    assert(sm == 1); // popcount of 1.
    const int mask = dim_num - 1;

    // Initialize the host input vectors
    for (int i = 0; i < idx_num; ++i)
    {
       h_A[i] = xorshift_hash(i) & mask;
       assert(h_A[i] < dim_num);
    }

    for (int i = 0; i < dim_num; ++i){
      h_B[i] = 5*i + 1;
    }


    // Allocate the device input vector A
    int *d_A = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_A, idx_size));

    // Allocate the device input vector B
    int *d_B = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_B, dim_size));

    // Allocate the device output vector C
    int *d_C = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_C, idx_size));


    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy idx from the host memory to the CUDA device\n");
    cudaCheckErrors(hipMemcpy(d_A, h_A, idx_size, hipMemcpyHostToDevice));
    cudaCheckErrors(hipMemcpy(d_B, h_B, dim_size, hipMemcpyHostToDevice));

    using namespace std::chrono;
    const int threadsPerBlock = 256; // try tuning this... no?
    const int blocksPerGrid = (idx_size + threadsPerBlock - 1) / threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    // Launch the Vector Add CUDA Kernel
    auto start = high_resolution_clock::now();    
    vectorGather<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, idx_num);
    cudaCheckErrors(hipDeviceSynchronize());
    auto end   = high_resolution_clock::now();
    auto diff = duration_cast<milliseconds>(end - start).count();
    cudaCheckErrors();
    
    cudaCheckErrors(hipMemcpy(h_C, d_C, idx_size, hipMemcpyDeviceToHost));
    printf("kernel runtime: %ld ms\n", diff);
    
    // Verify that the result vector is correct
    for (int i = 0; i < idx_num; ++i)
    {
      if (h_C[i] != h_A[i]*5+1)
        {
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED.\n");
    return 0;
}

