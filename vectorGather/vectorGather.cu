#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <chrono>
#include <functional>
#include <hip/hip_runtime_api.h>
#include <cassert>

inline int xorshift_hash(int x) {
    x ^= x >> 12; // a
    x ^= x << 25; // b
    x ^= x >> 27; // c
    return ((unsigned int)x) * 213338717U;
}


/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// __global__ void
// vectorAdd(const float *A, const float *B, float *C, int numElements)
// {
//     int i = blockDim.x * blockIdx.x + threadIdx.x;

//     if (i < numElements)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

__global__ void
vectorGather(const int * __restrict__ index_col, const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      output[i] = dimension_col[index_col[i]];
    }
}


// used as control
__global__ void
vectorCopy(const int * __restrict__ index_col, const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      output[i] = index_col[i];
    }
}


__global__ void
vectorGatherNoInput(const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len, int mask)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      auto x = i;
      auto a = x ^ (x >> 12); 
      auto b = a ^ (a << 25);
      auto c = b ^ (b >> 27);
      
      auto d  = ((unsigned int)c) * 213338717U;
      auto idx = d & mask;
      output[i] = dimension_col[idx];
    }
}

// basically CPU + write, to see how much is probably random.
__global__ void
vectorCopyNoInput(const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len, int mask)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      auto x = i;
      auto a = x ^ (x >> 12); 
      auto b = a ^ (a << 25);
      auto c = b ^ (b >> 27);
      
      auto d  = ((unsigned int)c) * 213338717U;
      auto idx = d & mask;
      output[i] = idx;
    }
}


#define cudaCheckErrors($call)                     \
    do { \
      hipError_t err = hipGetLastError(); \
      if (err != hipSuccess){\
        fprintf(stderr, "WARNING: Error was already set before call: (%s at %s:%d)\n", \
                hipGetErrorString(err),                       \
                __FILE__, __LINE__); \
      }\
      $call;                                  \
      err = hipGetLastError(); \
      if (err != hipSuccess) { \
        fprintf(stderr, "Fatal error: (%s at %s:%d)\n", \
                hipGetErrorString(err),                \
                __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
      } \
    } while (0)


/**
 * Host main routine
 */
int
main(void)
{
  printf("FYI:\ncuda cpuDeviceId: %d\n", hipCpuDeviceId);
  auto dev = 0;
  cudaCheckErrors(hipSetDevice(dev));
  hipDeviceProp_t deviceProp;
  cudaCheckErrors(hipGetDeviceProperties(&deviceProp, dev));

  printf("some device %d properties:\n",dev);
  printf("concurrent kernels %d\n",deviceProp.concurrentKernels);
  printf("device overlap %d\n",deviceProp.deviceOverlap);
  printf("max threads per block %d\n",deviceProp.maxThreadsPerBlock);
  printf("warp size %d\n",deviceProp.warpSize);
  printf("regs per block %d\n",deviceProp.regsPerBlock);
  
  // TODO: enable this in order to try mapped memory (vs streaming)
  // hipSetDeviceFlags(hipDeviceMapHost);
  
    // Print the vector length to be used, and compute its size
    const int G = 30;
    const int M = 20;
    //const int K = 10;

    
    size_t idx_size = 1U << G;
    size_t dim_size = 512U << M;
    
    size_t idx_num = idx_size / sizeof(int);
    size_t dim_num = dim_size / sizeof(int);
    
    printf("[Gather of %lu indices into a table of %lu locations]\n", idx_num, dim_num);

    // Allocate the host input vector A
    int *h_A = nullptr;
    cudaCheckErrors(hipMallocManaged(&h_A, idx_size));

    // Allocate the host input vector B
    int *h_B = nullptr;
    cudaCheckErrors(hipMallocManaged(&h_B, dim_size));
    
    int sm = __builtin_popcountl (dim_num);
    assert(sm == 1); // popcount of 1.
    const int mask = dim_num - 1;

    // Initialize the host input vectors
    for (int i = 0; i < idx_num; ++i)
    {
       h_A[i] = xorshift_hash(i) & mask;
       assert(h_A[i] < dim_num);
    }

    for (int i = 0; i < dim_num; ++i){
      h_B[i] = 5*i + 1;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    cudaCheckErrors(hipMalloc(&d_A, idx_size));

    // Allocate the device input vector B
    int *d_B = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_B, dim_size));

    // Allocate the device output vector C
    int *d_C = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_C, idx_size));

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy idx from the host memory to the CUDA device\n");
    cudaCheckErrors(hipMemcpy(d_A, h_A, idx_size, hipMemcpyHostToDevice));
    cudaCheckErrors(hipMemcpy(d_B, h_B, dim_size, hipMemcpyHostToDevice));

    using namespace std::chrono;
    const int threadsPerBlock = 256; // tried tuning. 
    const int blocksPerGrid = (idx_size + threadsPerBlock - 1) / threadsPerBlock; 
    printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);

    {
      auto start = high_resolution_clock::now();    
      //vectorCopy<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, idx_num);
      //vectorCopyNoInput<<<blocksPerGrid, threadsPerBlock>>>(d_B, d_C, idx_num, mask);
      vectorGatherNoInput<<<blocksPerGrid, threadsPerBlock>>>(d_B, d_C, idx_num, mask);
      cudaCheckErrors(hipDeviceSynchronize());
      auto end   = high_resolution_clock::now();
      auto diff = duration_cast<milliseconds>(end - start).count();
      cudaCheckErrors();
      printf("kernel runtime: %ld ms\n", diff);
    }


    // Allocate the host output vector C for checking.
    int *h_C = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_C, idx_size));
    cudaCheckErrors(hipMemcpy(h_C, d_C, idx_size, hipMemcpyDeviceToHost));
    
    // Verify that the result vector is correct
    for (int i = 0; i < idx_num; ++i)
    {
      if (h_C[i] != h_A[i]*5+1)
        {
          fprintf(stderr, "Result verification failed at element %d!\n", i);
          exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED.\n");
    return 0;
}

