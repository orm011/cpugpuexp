#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <benchmark/benchmark.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <chrono>
#include <functional>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <iostream>

using namespace std;

// TODO: enable this in order to try mapped memory (vs streaming)
// hipSetDeviceFlags(hipDeviceMapHost);
// Print the vector length to be used, and compute its size
constexpr int G = 30;
constexpr int M = 20;
constexpr int K = 10;

constexpr int kWarpSize = 32;
constexpr int kNumSM = 24; // gpu specific.

inline int xorshift_hash(int x) {
    x ^= x >> 12; // a
    x ^= x << 25; // b
    x ^= x >> 27; // c
    return ((unsigned int)x) * 213338717U;
}


// aka.1024. worked slightly better.
// it means each of the 4 exec units has 8 threads it can try to schedule
// and can hide latency up to 8x of

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// __global__ void
// vectorAdd(const float *A, const float *B, float *C, int numElements)
// {
//     int i = blockDim.x * blockIdx.x + threadIdx.x;

//     if (i < numElements)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

enum struct Variant {
  Mat,
  NoMat,
  OnlyMat,
  NaiveMemcpy,
  CudaMemcpy,
  CudaMemset,
  MAXVARIANT // do not use.
};


enum ILP {
  ilp1 = 1,
  ilp2 = 2,
  ilp4 = 4,
  ilp8 = 8,
  ilp16 = 16,
  ilp32 = 32,
};


struct ActiveThreads {
private:
  int val;

public:
  explicit ActiveThreads(int v) : val(v){}
  operator int(){ return val; }
};


template <Variant variant, ILP ilp, int ActiveThreads> __global__ void
templateKernel(const int * __restrict__ index_col,
               const int *__restrict__ dimension_col,
               int * __restrict__ output,
               int idx_len,
               int idx_domain)
{
  static_assert(ActiveThreads <= kWarpSize, "limit");
  static_assert(ActiveThreads > 0, "limit");
  static_assert((ActiveThreads - 1 & ActiveThreads) == 0, "power of 2"); // power of 2
  constexpr uint32_t active_mask =  (uint32_t)(((1UL << ActiveThreads)) - 1);
  
  uint32_t lanebit = 0;
  asm("mov.u32 %0, %lanemask_eq;" : "=r"(lanebit));
  
  // mapping block to data
  constexpr int64_t warpFraction = kWarpSize / ActiveThreads;
  int64_t blockSize = (ilp * blockDim.x)/warpFraction;
  int64_t blockStart = blockSize * blockIdx.x;
  int64_t blockEnd = blockStart + blockSize;

  // mapping warp to data
  constexpr int64_t dataPerWarp = ilp * ActiveThreads;
  int64_t warpNo = threadIdx.x / kWarpSize;
  int64_t warpOffset = blockStart + warpNo * dataPerWarp;

  
  if (lanebit & active_mask) {
   const auto lane =  __ffs(lanebit) - 1;
  const auto offset = warpOffset + lane;
  const auto unknown_variant = 0;
  const auto mask = idx_domain - 1;

  int tmp[ilp];

  // most blocks.
  if (blockEnd <= idx_len) {
    
  // load phase
  for (int g = 0; g < ilp; ++g) {
    auto item = offset + g*ActiveThreads;
        switch(variant) {
        case Variant::NaiveMemcpy:
          tmp[g] = index_col[item];
          break;
        case Variant::Mat:
          tmp[g] = dimension_col[index_col[item]];
          break;
        case Variant::OnlyMat:
          tmp[g] = 5*index_col[item] + 1;
          break;
        case Variant::NoMat:
          {
            auto x = item;
            auto a = x ^ (x >> 12); 
            auto b = a ^ (a << 25);
            auto c = b ^ (b >> 27);
            
            auto d  = ((unsigned int)c) * 213338717U;
            auto idx = d & mask;
            tmp[g] = dimension_col[idx];
            break;
          }
        default:
          assert(unknown_variant);
        }
  }
    // use phase
    for (int g = 0; g < ilp; ++g) {
      auto item = offset + g*ActiveThreads;
      switch(variant){
      default:
        output[item] = tmp[g];
      }
    }

  } else { // used only for the last thread block.
    for (int g = 0; g < ilp; ++g) {
    auto item = offset + g*ActiveThreads;
    
    if (item < idx_len)
      {

        switch(variant){
        case Variant::NaiveMemcpy:
          output[item] = index_col[item];
          break;
        case Variant::Mat:
          output[item] = dimension_col[index_col[item]];
          break;
        case Variant::OnlyMat:
          output[item] = 5*index_col[item] + 1;
          break;
        case Variant::NoMat:
          {
            auto x = item;
            auto a = x ^ (x >> 12); 
            auto b = a ^ (a << 25);
            auto c = b ^ (b >> 27);
            
            auto d  = ((unsigned int)c) * 213338717U;
            auto idx = d & mask;
            output[item] = dimension_col[idx];
            break;
          }
        default:
          assert(unknown_variant);
        }
      }
  }
  }

  }
}


#define cudaCheckErrors($call)                     \
    do { \
      hipError_t err = hipGetLastError(); \
      if (err != hipSuccess){\
        fprintf(stderr, "WARNING: Error was already set before call: (%s at %s:%d)\n", \
                hipGetErrorString(err),                       \
                __FILE__, __LINE__); \
      }\
      $call;                                  \
      err = hipGetLastError(); \
      if (err != hipSuccess) { \
        fprintf(stderr, "Fatal error: (%s at %s:%d)\n", \
                hipGetErrorString(err),                \
                __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
      } \
    } while (0)


using KernelT = void(const int *, const int *, int *, int, int);
template <Variant variant, ILP ilp, int ThreadsPerBlock, int ActiveThreads>
void GPU_BM(benchmark::State& state)
{
  static_assert(int32_t(variant) < int32_t(Variant::MAXVARIANT), "invalid variant");
  //cerr << "running bench again" << endl;
  //printf("FYI:\ncuda cpuDeviceId: %d\n", hipCpuDeviceId);
  int64_t idx_size = state.range(0);
  int64_t dim_size = state.range(1);
  int64_t idx_num = idx_size / sizeof(int);
  int64_t dim_num = dim_size / sizeof(int);

  //auto dev = 0;
  //cudaCheckErrors(hipSetDevice(dev));
  //hipDeviceProp_t deviceProp;
  //cudaCheckErrors(hipGetDeviceProperties(&deviceProp, dev));

  // printf("some device %d properties:\n",dev);
  // printf("concurrent kernels %d\n",deviceProp.concurrentKernels);
  // printf("device overlap %d\n",deviceProp.deviceOverlap);
  // printf("max threads per block %d\n",deviceProp.maxThreadsPerBlock);
  // printf("warp size %d\n",deviceProp.warpSize);
  // printf("regs per block %d\n",deviceProp.regsPerBlock);
  // printf("[Gather of %lu indices into a table of %lu locations]\n", idx_num, dim_num);

    // Allocate the host input vector A
    int *h_A = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_A, idx_size));

    // Allocate the host input vector B
    int *h_B = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_B, dim_size));
    
    int sm = __builtin_popcountl (dim_num);
    assert(sm == 1); // popcount of 1.
    const int mask = dim_num - 1;

    // Initialize the host input vectors
    for (int i = 0; i < idx_num; ++i)
    {
       h_A[i] = xorshift_hash(i) & mask;
       assert(h_A[i] < dim_num);
    }

    for (int i = 0; i < dim_num; ++i){
      h_B[i] = 5*i + 1;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    cudaCheckErrors(hipMalloc(&d_A, idx_size));

    // Allocate the device input vector B
    int *d_B = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_B, dim_size));

    // Allocate the device output vector C
    int *d_C = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_C, idx_size));
    

    int itemsPerBlock = -1;
    int blocksPerGrid = -1;
    
    cudaCheckErrors(hipMemcpy(d_B, h_B, dim_size, hipMemcpyHostToDevice));
    cudaCheckErrors(hipMemcpy(d_A, h_A, idx_size, hipMemcpyHostToDevice));

    KernelT* kernel = nullptr;
    switch (variant){
    case Variant::CudaMemcpy:
    case Variant::CudaMemset:
      break;
    default:{
      kernel=templateKernel<variant, ilp, ActiveThreads>;
      itemsPerBlock = (ilp * ThreadsPerBlock * ActiveThreads)/kWarpSize;
      blocksPerGrid = (idx_size + itemsPerBlock - 1) / itemsPerBlock;
      fprintf(stderr,
              "Variant: %d\n"
              "ILP: %d\n"
              "Active threads per warp: %d\n"
              "Threads per block: %d\n"
              "Blocks per SM: %d\n"
              "Remainder blocks: %d\n"
              "Remainder threads: %d\n",
              int(variant),
              ilp,
              ActiveThreads,
              ThreadsPerBlock,
              blocksPerGrid / kNumSM,
              blocksPerGrid % kNumSM,
              2048 % ThreadsPerBlock);

    }
    }
    

    

    while (state.KeepRunning()){
        switch (variant) {
        case Variant::CudaMemcpy:
          hipMemcpy(d_C, d_A, idx_size, hipMemcpyDeviceToDevice);
          break;
        case Variant::CudaMemset:
          hipMemset(d_C, 0xf, idx_size);
          break;
        default:
          kernel<<<blocksPerGrid, ThreadsPerBlock>>>(d_A, d_B, d_C, idx_num, dim_num);
          break;
        }
        
        hipDeviceSynchronize();
    }

    state.SetItemsProcessed(int64_t(state.iterations())*int64_t(idx_num));

    switch(variant){
    case Variant::CudaMemcpy:
    case Variant::NaiveMemcpy:
      state.SetBytesProcessed(int64_t(state.iterations())*
                              int64_t(idx_size * 2)); // read write
      break;
    case Variant::CudaMemset:
      state.SetBytesProcessed(int64_t(state.iterations())*
                              int64_t(idx_size)); // read write
      break;
    default:
      break;
    }
    
    // Allocate the host output vector C for checking.
    int *h_C = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_C, idx_size));
    cudaCheckErrors(hipMemcpy(h_C, d_C, idx_size, hipMemcpyDeviceToHost));

    // Verify that the result vector is correct
    switch (variant){
    case Variant::CudaMemcpy:
    case Variant::NaiveMemcpy:
      { 
        for (int i = 0; i < idx_num; ++i) {
          if (h_C[i] != h_A[i]) {
            fprintf(stderr, "ERROR. memcpy verification failed at position %d: h_C=%d but h_A=%d\n", i, h_C[i], h_A[i]);
            break; // free memory
          }
        }
        break;
      }
    case Variant::CudaMemset:
      {
        for (int i = 0; i < idx_num; ++i){
          if (h_C[i] != 0x0f0f0f0f){
            fprintf(stderr,  "ERROR. memset verification failed\n");
            break; // free memory
          }
        }
        break;
      }
    default:
      {// mbold red text
        for (int i = 0; i < idx_num; ++i) {
            if (h_C[i] != h_A[i]*5+1) {
              fprintf(stderr, "\033[1;31mERROR.\033[0mgather verification failed at position %d: h_C=%d but h_A=%d and hA*5 + 1 = %d\n", i, h_C[i], h_A[i], h_A[i]*5+ 1);
              break; // free memory
            }
        }
        break; 
      }
    }

    cudaCheckErrors(hipHostFree(h_A));
    cudaCheckErrors(hipHostFree(h_B));
    cudaCheckErrors(hipHostFree(h_C));
    cudaCheckErrors(hipFree(d_A));
    cudaCheckErrors(hipFree(d_B));
    cudaCheckErrors(hipFree(d_C));
    //printf("Test PASSED.\n");
}

#define TPB(n) n
#define ATh(n) n
#define ILP(n) (ILP::ilp##n)

// BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(1), TPB(1024), ATh(32))
// ->RangeMultiplier(2)
// ->Ranges({{1<<G, 1<<G}, {1 << K, 1 << G}})
// ->Unit(benchmark::kMillisecond); 

// BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(1), TPB(1024), ATh(32)) // actually does write output for now..
// ->RangeMultiplier(2)
// ->Ranges({{1<<G, 1<<G}, {32 << M, 256 << M}})
// ->Unit(benchmark::kMillisecond);

// BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(2), TPB(1024), ATh(32)) // actually does write output for now..
// ->RangeMultiplier(2)
// ->Ranges({{1<<G, 1<<G}, {32 << M, 256 << M}})
// ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(32)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);

// BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(1), TPB(1024), ATh(8)) // actually does write output for now..
// ->RangeMultiplier(2)
// ->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
// ->Unit(benchmark::kMillisecond);

// BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(2), TPB(1024), ATh(8)) // actually does write output for now..
// ->RangeMultiplier(2)
// ->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
// ->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(16)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(8)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(4)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(2)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::Mat, ILP(4), TPB(1024), ATh(1)) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {32 << M, 32 << M}})
->Unit(benchmark::kMillisecond);


// BENCHMARK_TEMPLATE(GPU_BM, Variant::OnlyMat, ILP(1), TPB(1024), ATh(32))  // dim should be irrelevant
// ->Args({1 << G, 1 <<K})->Args({1<<G, 1<<G})
// ->Unit(benchmark::kMillisecond); 

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(1), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(2), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(4), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(8), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(16), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(8), TPB(512), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(16), TPB(512), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(16), TPB(512), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(8), TPB(256), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(16), TPB(256), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::NaiveMemcpy, ILP(32), TPB(256), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_TEMPLATE(GPU_BM, Variant::CudaMemcpy, ILP(1), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, Variant::CudaMemset, ILP(1), TPB(1024), ATh(32)) // dim should be irrelevant
->Args({1 << G, 1 << K})
->Unit(benchmark::kMillisecond);


BENCHMARK_MAIN();