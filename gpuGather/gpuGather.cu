#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>
#include <benchmark/benchmark.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <chrono>
#include <functional>
#include <hip/hip_runtime_api.h>
#include <cassert>
#include <iostream>

using namespace std;

// TODO: enable this in order to try mapped memory (vs streaming)
// hipSetDeviceFlags(hipDeviceMapHost);
// Print the vector length to be used, and compute its size
const int G = 30;
//const int M = 20;
const int K = 10;

inline int xorshift_hash(int x) {
    x ^= x >> 12; // a
    x ^= x << 25; // b
    x ^= x >> 27; // c
    return ((unsigned int)x) * 213338717U;
}

const int kDefaultTpB = 4*32*8; 
// aka.1024. worked slightly better.
// it means each of the 4 exec units has 8 threads it can try to schedule
// and can hide latency up to 8x of

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
// __global__ void
// vectorAdd(const float *A, const float *B, float *C, int numElements)
// {
//     int i = blockDim.x * blockIdx.x + threadIdx.x;

//     if (i < numElements)
//     {
//         C[i] = A[i] + B[i];
//     }
// }

enum Variant {
  Mat,
  NoMat,
  OnlyMat,
  OnlyWrite,
  MAXVARIANT // do not use.
};

__global__ void
gpuMat(const int * __restrict__ index_col, const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len, int idx_domain)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      output[i] = dimension_col[index_col[i]];
    }
}

// used as control
__global__ void
gpuOnlyMat(const int * __restrict__ index_col, const int *__restrict__, int * __restrict__ output, int idx_len, int idx_domain)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < idx_len)
    {
      output[i] = 5*index_col[i] + 1;
    }
}


__global__ void
gpuNoMat(const int *__restrict__ , const int * __restrict__ dimension_col, int * __restrict__ output, int idx_len, int idx_domain)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    const auto mask = idx_domain - 1;
    
    if (i < idx_len)
    {
      auto x = i;
      auto a = x ^ (x >> 12); 
      auto b = a ^ (a << 25);
      auto c = b ^ (b >> 27);
      
      auto d  = ((unsigned int)c) * 213338717U;
      auto idx = d & mask;
      output[i] = dimension_col[idx];
    }
}

// basically CPU + write, to see how much is probably random.
__global__ void
gpuOnlyWrite(const int* __restrict__ , const int *__restrict__ dimension_col, int * __restrict__ output, int idx_len, int idx_domain)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    const auto mask = idx_domain - 1;
    if (i < idx_len)
    {
      auto x = i;
      auto a = x ^ (x >> 12); 
      auto b = a ^ (a << 25);
      auto c = b ^ (b >> 27);
      
      auto d  = ((unsigned int)c) * 213338717U;
      auto idx = d & mask;
      output[i] = 5*idx +1;
    }
}

#define cudaCheckErrors($call)                     \
    do { \
      hipError_t err = hipGetLastError(); \
      if (err != hipSuccess){\
        fprintf(stderr, "WARNING: Error was already set before call: (%s at %s:%d)\n", \
                hipGetErrorString(err),                       \
                __FILE__, __LINE__); \
      }\
      $call;                                  \
      err = hipGetLastError(); \
      if (err != hipSuccess) { \
        fprintf(stderr, "Fatal error: (%s at %s:%d)\n", \
                hipGetErrorString(err),                \
                __FILE__, __LINE__); \
        fprintf(stderr, "*** FAILED - ABORTING\n"); \
        exit(1); \
      } \
    } while (0)


using KernelT = void(const int *, const int *, int *, int, int);
template <Variant variant, int ThreadsPerBlock>
void GPU_BM(benchmark::State& state)
{
  static_assert(variant < MAXVARIANT, "invalid variant");
  //cerr << "running bench again" << endl;
  //printf("FYI:\ncuda cpuDeviceId: %d\n", hipCpuDeviceId);
  int64_t idx_size = state.range(0);
  int64_t dim_size = state.range(1);
  int64_t idx_num = idx_size / sizeof(int);
  int64_t dim_num = dim_size / sizeof(int);

  //auto dev = 0;
  //cudaCheckErrors(hipSetDevice(dev));
  //hipDeviceProp_t deviceProp;
  //cudaCheckErrors(hipGetDeviceProperties(&deviceProp, dev));

  // printf("some device %d properties:\n",dev);
  // printf("concurrent kernels %d\n",deviceProp.concurrentKernels);
  // printf("device overlap %d\n",deviceProp.deviceOverlap);
  // printf("max threads per block %d\n",deviceProp.maxThreadsPerBlock);
  // printf("warp size %d\n",deviceProp.warpSize);
  // printf("regs per block %d\n",deviceProp.regsPerBlock);
  // printf("[Gather of %lu indices into a table of %lu locations]\n", idx_num, dim_num);

    // Allocate the host input vector A
    int *h_A = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_A, idx_size));

    // Allocate the host input vector B
    int *h_B = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_B, dim_size));
    
    int sm = __builtin_popcountl (dim_num);
    assert(sm == 1); // popcount of 1.
    const int mask = dim_num - 1;

    // Initialize the host input vectors
    for (int i = 0; i < idx_num; ++i)
    {
       h_A[i] = xorshift_hash(i) & mask;
       assert(h_A[i] < dim_num);
    }

    for (int i = 0; i < dim_num; ++i){
      h_B[i] = 5*i + 1;
    }

    // Allocate the device input vector A
    int *d_A = NULL;
    cudaCheckErrors(hipMalloc(&d_A, idx_size));

    // Allocate the device input vector B
    int *d_B = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_B, dim_size));

    // Allocate the device output vector C
    int *d_C = NULL;
    cudaCheckErrors(hipMalloc((void **)&d_C, idx_size));
    
    const int threadsPerBlock = ThreadsPerBlock;
    const int blocksPerGrid = (idx_size + threadsPerBlock - 1) / threadsPerBlock;
    fprintf(stderr, "NB. threads per block = %d. num blocks = %d. blocks per sm = %d\n", threadsPerBlock, blocksPerGrid, blocksPerGrid/24);
    
    cudaCheckErrors(hipMemcpy(d_B, h_B, dim_size, hipMemcpyHostToDevice));
    cudaCheckErrors(hipMemcpy(d_A, h_A, idx_size, hipMemcpyHostToDevice));

    KernelT* kernel;
    switch (variant){
    case Mat:
      kernel=gpuMat;
      break;
    case OnlyMat:
      kernel=gpuOnlyMat;
      break;
    case OnlyWrite:
      kernel=gpuOnlyWrite;
      break;
    case NoMat:
      kernel=gpuNoMat;
      break;
    default:
      assert(false && "unknown variant");
    }
    
    while (state.KeepRunning())
    {
      kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, idx_num, dim_num);
      hipDeviceSynchronize();
    }

    if (variant == OnlyMat){
      state.SetBytesProcessed(int64_t(state.iterations()) *
                              int64_t(idx_size * 2));
    }
    
    // Allocate the host output vector C for checking.
    int *h_C = nullptr;
    cudaCheckErrors(hipHostMalloc(&h_C, idx_size));
    cudaCheckErrors(hipMemcpy(h_C, d_C, idx_size, hipMemcpyDeviceToHost));
    
    // Verify that the result vector is correct
    for (int i = 0; i < idx_num; ++i)
      {
        if (h_C[i] != h_A[i]*5+1)
          {
            fprintf(stdout, "Result verification first failed at element %d!. has: %d. expected: %d\n", i, h_C[i], h_A[i]*5 + 1);
            break;
          }
      }

    cudaCheckErrors(hipHostFree(h_A));
    cudaCheckErrors(hipHostFree(h_B));
    cudaCheckErrors(hipHostFree(h_C));
    cudaCheckErrors(hipFree(d_A));
    cudaCheckErrors(hipFree(d_B));
    cudaCheckErrors(hipFree(d_C));
    //printf("Test PASSED.\n");
}


BENCHMARK_TEMPLATE(GPU_BM, Mat, kDefaultTpB)
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {1 << K, 1 << G}})
->Unit(benchmark::kMillisecond); 

BENCHMARK_TEMPLATE(GPU_BM, NoMat, kDefaultTpB) // actually does write output for now..
->RangeMultiplier(2)
->Ranges({{1<<G, 1<<G}, {1 << K, 1 << G}})
->Unit(benchmark::kMillisecond);

BENCHMARK_TEMPLATE(GPU_BM, OnlyMat, kDefaultTpB)  // dim should be irrelevant
->Args({1 << G, 1 <<K})->Args({1<<G, 1<<G})
->Unit(benchmark::kMillisecond); 

BENCHMARK_TEMPLATE(GPU_BM, OnlyWrite, kDefaultTpB) // dim should be irrelevant
->Args({1 << G, 1 <<K})->Args({1<<G, 1<<G})
->Unit(benchmark::kMillisecond); 

BENCHMARK_MAIN();